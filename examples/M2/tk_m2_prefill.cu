#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <math.h>
#include <assert.h>
//#include <mma_AB.h>
#include <hip/hip_runtime_api.h>
#include <mma.h>
#include <hip/hip_fp16.h>
using namespace nvcuda;

# include "../../src/kittens.cuh"
# include "../../src/common/pyutils/torch_helpers.cuh"

// **** ASYNC INCLUDE *****
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

using namespace kittens;


template <typename H, typename T>
__global__
void prefill_whole_loop_ker(
        const int NC, const int CS, const int HF, const int HF_prime,
        T* __W1, T* __W2,
        const T* __XA, const T* __XB, const T* __XC,
        T* __Output
) {
    H *_W1       = reinterpret_cast<H*>(__W1) + blockIdx.x * (HF * HF_prime);
    H *_W2       = reinterpret_cast<H*>(__W2) + blockIdx.x * (HF_prime * HF);
    const H *_XA       = reinterpret_cast<const H*>(__XA) + blockIdx.x * (NC * CS * HF);
    const H *_XB       = reinterpret_cast<const H*>(__XB) + blockIdx.x * (NC * CS * HF);
    const H *_XC       = reinterpret_cast<const H*>(__XC) + blockIdx.x * (NC * CS * HF);
    H *_Output = reinterpret_cast<H*>(__Output) + blockIdx.x * (NC * CS * HF);

    rt_bf<4, 16, kittens::ducks::rt_layout::col> W1_col_reg;
    rt_fl<4, 16> delta_W1_fl_reg;
    rt_bf<4, 16> delta_W1_reg;

    rt_bf<16, 4, kittens::ducks::rt_layout::col> W2_col_reg;
    rt_fl<16, 4> delta_W2_fl_reg;
    rt_bf<16, 4> delta_W2_reg;

    rt_bf<1, 4> XA_reg;
    rt_bf<1, 4> XB_reg;
    rt_bf<1, 4> XC_reg;

    rt_fl<1, 16> Z1_fl_reg;
    rt_bf<1, 16> Z1_reg;
    rt_bf<1, 16> dl_dZ1_reg;
    rt_fl<1, 16> dl_dZ1_fl_reg;

    rt_fl<1, 16> Z1_bar_term_1_fl_reg;
    rt_bf<1, 16> Z1_bar_term_1_reg;
    rt_fl<1, 16> Z1_bar_term_2_fl_reg;
    rt_bf<1, 16> Z1_bar_term_2_reg;

    rt_fl<1, 4> Z2_fl_reg;
    rt_bf<1, 4> dl_dZ2_reg;

    rt_fl<1, 4> Z2_bar_term_1_fl_reg;
    rt_bf<1, 4> Z2_bar_term_1_reg;
    rt_fl<1, 4> Z2_bar_term_2_fl_reg;
    rt_bf<1, 4> Z2_bar_term_2_reg;

    rt_fl<1, 1> Attn_fl_reg;
    rt_bf<1, 1> Attn_reg;

    load(W1_col_reg, _W1, W1_col_reg.cols);
    load(W2_col_reg, _W2, W2_col_reg.cols);

    for (int i = 0; i < NC; i++) {
        // Forward
        load(XB_reg, _XB + i * CS * HF, XB_reg.cols);  // [K,f]
        zero(Z1_fl_reg);  // [K,4f]
        mma_AB(Z1_fl_reg, XB_reg, W1_col_reg, Z1_fl_reg); // [K,f]r, [f,4f]c -> [K,4f]r
        copy(Z1_reg, Z1_fl_reg);
        zero(Z2_fl_reg); // [K,f]
        mma_AB(Z2_fl_reg, Z1_reg, W2_col_reg, Z2_fl_reg); // [K,4f]r, [4f,f]c -> [K,f]r

        // dl_dZ2
        load(XA_reg, _XA + i * CS * HF, XA_reg.cols);  // [K,f]
        copy(dl_dZ2_reg, Z2_fl_reg);
        sub(dl_dZ2_reg, dl_dZ2_reg, XA_reg);  // [K,f]

        // delta W2
//        rt_bf<1, 16, ducks::rt_layout::col> &Z1_col_reg = swap_layout_inplace(Z1_reg);  // [K,f] r -> c
        rt_bf<1, 16, ducks::rt_layout::col> Z1_col_reg;
        swap_layout(Z1_col_reg, Z1_reg);
//        rt_bf<1, 4, ducks::rt_layout::col> &dl_dZ2_col_reg = swap_layout_inplace(dl_dZ2_reg);  // [K,f] r -> c
        rt_bf<1, 4, ducks::rt_layout::col> dl_dZ2_col_reg;
        swap_layout(dl_dZ2_col_reg, dl_dZ2_reg);
        zero(delta_W2_fl_reg);
        mma_AtB(delta_W2_fl_reg, Z1_col_reg, dl_dZ2_col_reg, delta_W2_fl_reg);  // ([K,4f]c).t @ [K,f]c -> [4f,f]r
        copy(delta_W2_reg, delta_W2_fl_reg);
//        rt_bf<16, 4, ducks::rt_layout::col> &delta_W2_col_reg = swap_layout_inplace(delta_W2_reg);
        rt_bf<16, 4, ducks::rt_layout::col> delta_W2_col_reg;
        swap_layout(delta_W2_col_reg, delta_W2_reg);

        // dl_dZ1
        zero(dl_dZ1_fl_reg);
//        rt_bf<16, 4, kittens::ducks::rt_layout::row> &W2_reg = swap_layout_inplace(W2_col_reg);
        rt_bf<16, 4, kittens::ducks::rt_layout::row> W2_reg;  // TODO: alternatively, swap in-place twice
        swap_layout(W2_reg, W2_col_reg);
        mma_ABt(dl_dZ1_fl_reg, dl_dZ2_reg, W2_reg, dl_dZ1_fl_reg);  // [K,f]r @ [4f,f]r.t -> [K,4f]r
        copy(dl_dZ1_reg, dl_dZ1_fl_reg);

        // delta W1
//        rt_bf<1, 4, ducks::rt_layout::col> &XB_col_reg = swap_layout_inplace(XB_reg);  // [K,f] r -> c
        rt_bf<1, 4, ducks::rt_layout::col> XB_col_reg;
        swap_layout(XB_col_reg, XB_reg);
//        rt_bf<1, 16, ducks::rt_layout::col> &dl_dZ1_col_reg = swap_layout_inplace(dl_dZ1_reg);  // [K,4f] r -> c
        rt_bf<1, 16, ducks::rt_layout::col> dl_dZ1_col_reg;
        swap_layout(dl_dZ1_col_reg, dl_dZ1_reg);
        zero(delta_W1_fl_reg);
        mma_AtB(delta_W1_fl_reg, XB_col_reg, dl_dZ1_col_reg, delta_W1_fl_reg);  // ([K,f]c).t @ [K,4f]c -> [f,4f]r
        copy(delta_W1_reg, delta_W1_fl_reg);
//        rt_bf<4, 16, ducks::rt_layout::col> &delta_W1_col_reg = swap_layout_inplace(delta_W1_reg);
        rt_bf<4, 16, ducks::rt_layout::col> delta_W1_col_reg;
        swap_layout(delta_W1_col_reg, delta_W1_reg);

        // Attn1
        load(XC_reg, _XC + i * CS * HF, XC_reg.cols);  // [K,f]
        zero(Attn_fl_reg);  // [K,K]
        mma_ABt(Attn_fl_reg, XC_reg, XB_reg, Attn_fl_reg);  // [K,f]r @ [K,f]r.t -> [K,K]r
        copy(Attn_reg, Attn_fl_reg);
        make_causal(Attn_reg, Attn_reg, base_types::constants<bf16>::zero());

        // Z1_bar
        zero(Z1_bar_term_1_fl_reg);
        mma_AB(Z1_bar_term_1_fl_reg, XC_reg, W1_col_reg, Z1_bar_term_1_fl_reg);  // [K,f]r, [f,4f]c -> [K,4f]r
        copy(Z1_bar_term_1_reg, Z1_bar_term_1_fl_reg);

        zero(Z1_bar_term_2_fl_reg);
        mma_AB(Z1_bar_term_2_fl_reg, Attn_reg, dl_dZ1_col_reg, Z1_bar_term_2_fl_reg);  // [K,K]r, [K,f]c -> [K,f]r
        copy(Z1_bar_term_2_reg, Z1_bar_term_2_fl_reg);

        sub(Z1_bar_term_1_reg, Z1_bar_term_1_reg, Z1_bar_term_2_reg);  // cannot multiplex Z2_bar and Z2_bar_term_1_reg

        // Attn2
        zero(Attn_fl_reg);  // [K,K]
        mma_ABt(Attn_fl_reg, Z1_bar_term_1_reg, Z1_reg, Attn_fl_reg);  // [K,K]r, [K,f]r -> [K,f]r
        copy(Attn_reg, Attn_fl_reg);
        make_causal(Attn_reg, Attn_reg, base_types::constants<bf16>::zero());

        // Z2_bar
        zero(Z2_bar_term_1_fl_reg);
        mma_AB(Z2_bar_term_1_fl_reg, Z1_bar_term_1_reg, W2_col_reg, Z2_bar_term_1_fl_reg);
        copy(Z2_bar_term_1_reg, Z2_bar_term_1_fl_reg);

        zero(Z2_bar_term_2_fl_reg);
        mma_AB(Z2_bar_term_2_fl_reg, Attn_reg, dl_dZ2_col_reg, Z2_bar_term_2_fl_reg);
        copy(Z2_bar_term_2_reg, Z2_bar_term_2_fl_reg);

        sub(Z2_bar_term_1_reg, Z2_bar_term_1_reg, Z2_bar_term_2_reg);  // cannot multiplex Z2_bar and Z2_bar_term_1_reg

        // Store Output
        store(_Output + i * CS * HF, Z2_bar_term_1_reg, Z2_bar_term_1_reg.cols);

        // Updated W1, W2
        sub(W1_col_reg, W1_col_reg, delta_W1_col_reg);
        sub(W2_col_reg, W2_col_reg, delta_W2_col_reg);
    }

    store(_W1, W1_col_reg, W1_col_reg.cols);
    store(_W2, W2_col_reg, W2_col_reg.cols);
}


void
prefill_whole_loop
        (
                torch::Tensor W1, torch::Tensor W2,
                torch::Tensor XA, torch::Tensor XB, torch::Tensor XC,
                torch::Tensor Output,
                hipStream_t stream
        ) {
    auto batch = XA.size(0);
    auto head = XA.size(1);
    auto NC = XA.size(2);
    auto CS = XA.size(3);
    auto HF = XA.size(4);
    auto HF_prime = W1.size(3);  // [BS,NH,HF,HF_prime]

//    std::cout << "HF: " << HF << std::endl;
//    std::cout << "HF_prime: " << HF_prime << std::endl;

    using H = __hip_bfloat16;
    using T = c10::BFloat16;
    const int workers = 1;

    auto threads = workers * kittens::WARP_THREADS;

    prefill_whole_loop_ker<H, T><<<batch * head, threads, 0, stream>>>(
            NC, CS, HF, HF_prime,
            W1.data_ptr<T>(), W2.data_ptr<T>(),
            XA.data_ptr<T>(), XB.data_ptr<T>(), XC.data_ptr<T>(),
            Output.data_ptr<T>()
    );


}

