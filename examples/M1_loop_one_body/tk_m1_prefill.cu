#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <assert.h>
//#include <mma_AB.h>
#include <string>
#include <hip/hip_runtime_api.h>
#include <mma.h>
#include <hip/hip_fp16.h>
using namespace nvcuda;

# include "../../src/kittens.cuh"
# include "../../src/common/pyutils/torch_helpers.cuh"

// **** ASYNC INCLUDE *****
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

using namespace kittens;


template <typename H, typename T>
__global__
void prefill_loop_body_ker(
        int CS, int HF,
        T* __W1,
        const T* __XA, const T* __XB, const T* __XC,
        T* __Output
) {
    H *_W1       = reinterpret_cast<H*>(__W1) + blockIdx.x*(HF*HF);
    const H *_XA       = reinterpret_cast<const H*>(__XA) + blockIdx.x*(CS*HF);
    const H *_XB       = reinterpret_cast<const H*>(__XB) + blockIdx.x*(CS*HF);
    const H *_XC       = reinterpret_cast<const H*>(__XC) + blockIdx.x*(CS*HF);
    H *_Output = reinterpret_cast<H*>(__Output) + blockIdx.x*(CS*HF);

    /*********
    REGISTER
    **********/
    rt_bf<4, 4, kittens::ducks::rt_layout::col> W1_reg;
    rt_bf<1, 4> XA_reg;
    rt_bf<1, 4> XB_reg;
    rt_bf<1, 4> XC_reg;

    rt_fl<1, 4> Z1_fl_reg;
    rt_bf<1, 4> Z1_reg;

    rt_bf<1, 4> Output_reg;
    rt_fl<1, 4> Z1_bar_term_1_fl_reg;
    rt_bf<1, 4> Z1_bar_term_1_reg;
    rt_fl<1, 4> Z1_bar_term_2_fl_reg;
    rt_bf<1, 4> Z1_bar_term_2_reg;
    rt_fl<1, 1> Attn1_fl_reg;
    rt_bf<1, 1> Attn1_reg;


    load(W1_reg, _W1, W1_reg.cols);
    load(XB_reg, _XB, XB_reg.cols);
    load(XA_reg, _XA, XA_reg.cols);
    load(XC_reg, _XC, XC_reg.cols);

    zero(Z1_fl_reg);
    mma_AB(Z1_fl_reg, XB_reg, W1_reg, Z1_fl_reg); // [K,f] r, [f,f] c -> [K,f] r

    copy(Z1_reg, Z1_fl_reg);
    sub(Z1_reg, Z1_reg, XA_reg);

    rt_bf<1, 4, ducks::rt_layout::col> &Z1_col_reg = swap_layout_inplace(Z1_reg); // row-maj -> col-maj

    zero(Attn1_fl_reg);
    mma_ABt(Attn1_fl_reg, XC_reg, XB_reg, Attn1_fl_reg);  // [N,K] r, [M,K] r -> [N,M] r
    copy(Attn1_reg, Attn1_fl_reg);
    make_causal(Attn1_reg, Attn1_reg, base_types::constants<bf16>::zero());

    zero(Z1_bar_term_1_fl_reg);
    mma_AB(Z1_bar_term_1_fl_reg, XC_reg, W1_reg, Z1_bar_term_1_fl_reg); // [N,K] r, [K,M] c -> [N,M] r
    copy(Z1_bar_term_1_reg, Z1_bar_term_1_fl_reg);

    zero(Z1_bar_term_2_fl_reg);
    mma_AB(Z1_bar_term_2_fl_reg, Attn1_reg, Z1_col_reg, Z1_bar_term_2_fl_reg);  // [K,K] r, [K,f] c -> [K,f] r
    copy(Z1_bar_term_2_reg, Z1_bar_term_2_fl_reg);

    sub(Output_reg, Z1_bar_term_1_reg, Z1_bar_term_2_reg);

    store(_Output, Output_reg, Output_reg.cols);

    rt_bf<1, 4, kittens::ducks::rt_layout::col> &XB_col_reg = swap_layout_inplace(XB_reg);

    rt_fl<4, 4> W1_fl_reg;
    zero(W1_fl_reg);
    mma_AtB(W1_fl_reg, XB_col_reg, Z1_col_reg, W1_fl_reg);

    rt_bf<4, 4> W1_row_reg;
    copy(W1_row_reg, W1_fl_reg);
    rt_bf<4, 4, kittens::ducks::rt_layout::col> &W1_col_reg = swap_layout_inplace(W1_row_reg);

    sub(W1_reg, W1_reg, W1_col_reg);

    store(_W1, W1_reg, W1_reg.cols);

}

void
prefill_loop_body
        (
                torch::Tensor W1,
                torch::Tensor XA,
                torch::Tensor XB,
                torch::Tensor XC,
                torch::Tensor Output,
                hipStream_t stream
        ) {

    auto batch_mul_head = XA.size(0);
    auto cs    = XA.size(1);
    auto hf    = XA.size(2);

    using H = __hip_bfloat16;
    using T = c10::BFloat16;
    const int workers = 1;
    auto threads = workers * kittens::WARP_THREADS;

    prefill_loop_body_ker<H,T><<<batch_mul_head, threads, 0, stream>>>(
            cs, hf,
            W1.data_ptr<T>(),
            XA.data_ptr<T>(), XB.data_ptr<T>(), XC.data_ptr<T>(),
            Output.data_ptr<T>()
    );

    // CHECK_CUDA_ERROR(hipDeviceSynchronize());
}


template <typename H, typename T>
__global__
void prefill_whole_loop_ker(
        const int NC, const int CS, const int HF,
        T* __W1,
        const T* __XA, const T* __XB, const T* __XC,
        T* __Output
) {
    H *_W1       = reinterpret_cast<H*>(__W1) + blockIdx.x * (HF*HF);
    const H *_XA       = reinterpret_cast<const H*>(__XA) + blockIdx.x * (NC*CS*HF);
    const H *_XB       = reinterpret_cast<const H*>(__XB) + blockIdx.x * (NC*CS*HF);
    const H *_XC       = reinterpret_cast<const H*>(__XC) + blockIdx.x * (NC*CS*HF);
    H *_Output = reinterpret_cast<H*>(__Output) + blockIdx.x * (NC*CS*HF);

    rt_bf<4, 4, kittens::ducks::rt_layout::col> W1_reg;
    rt_bf<1, 4> XA_reg;
    rt_bf<1, 4> XB_reg;
    rt_bf<1, 4> XC_reg;

    rt_fl<1, 4> Z1_fl_reg;
    rt_bf<1, 4> Z1_reg;

    rt_bf<1, 4> Output_reg;
    rt_fl<1, 4> Z1_bar_term_1_fl_reg;
    rt_bf<1, 4> Z1_bar_term_1_reg;
    rt_fl<1, 4> Z1_bar_term_2_fl_reg;
    rt_bf<1, 4> Z1_bar_term_2_reg;
    rt_fl<1, 1> Attn1_fl_reg;
    rt_bf<1, 1> Attn1_reg;
    rt_fl<4, 4> W1_fl_reg;
    rt_bf<4, 4> W1_row_reg;

    load(W1_reg, _W1, W1_reg.cols);

    for (int i = 0; i < NC; i++) {

        // rt_bf<1, 4> XB_reg;
        load(XB_reg, _XB + i * CS * HF, XB_reg.cols);
        // rt_fl<1, 4> Z1_fl_reg;
        zero(Z1_fl_reg);
        mma_AB(Z1_fl_reg, XB_reg, W1_reg, Z1_fl_reg); // [K,f] r, [f,f] c -> [K,f] r

        // rt_bf<1, 4> XA_reg;
        load(XA_reg, _XA + i * CS * HF, XA_reg.cols);

        // rt_bf<1, 4> Z1_reg;
        copy(Z1_reg, Z1_fl_reg);
        sub(Z1_reg, Z1_reg, XA_reg);

        rt_bf<1, 4, ducks::rt_layout::col> &Z1_col_reg = swap_layout_inplace(Z1_reg);

        // rt_bf<1, 4> XC_reg;
        load(XC_reg, _XC + i * CS * HF, XC_reg.cols);
        // rt_fl<1, 1> Attn1_fl_reg;
        zero(Attn1_fl_reg);
        mma_ABt(Attn1_fl_reg, XC_reg, XB_reg, Attn1_fl_reg);

        // rt_bf<1, 1> Attn1_reg;
        copy(Attn1_reg, Attn1_fl_reg);
        make_causal(Attn1_reg, Attn1_reg, base_types::constants<bf16>::zero());

        // rt_fl<1, 4> Z1_bar_term_1_fl_reg;
        zero(Z1_bar_term_1_fl_reg);
        mma_AB(Z1_bar_term_1_fl_reg, XC_reg, W1_reg, Z1_bar_term_1_fl_reg); // [N,K] r, [K,M] c -> [N,M] r
        // rt_bf<1, 4> Z1_bar_term_1_reg;
        copy(Z1_bar_term_1_reg, Z1_bar_term_1_fl_reg);

        // rt_fl<1, 4> Z1_bar_term_2_fl_reg;
        zero(Z1_bar_term_2_fl_reg);
        mma_AB(Z1_bar_term_2_fl_reg, Attn1_reg, Z1_col_reg, Z1_bar_term_2_fl_reg);  // [K,K] r, [K,f] c -> [K,f] r
        // rt_bf<1, 4> Z1_bar_term_2_reg;
        copy(Z1_bar_term_2_reg, Z1_bar_term_2_fl_reg);

        // rt_bf<1, 4> Output_reg;
        sub(Output_reg, Z1_bar_term_1_reg, Z1_bar_term_2_reg);
        store(_Output + i * CS * HF, Output_reg, Output_reg.cols);
        rt_bf<1, 4, kittens::ducks::rt_layout::col> &XB_col_reg = swap_layout_inplace(XB_reg);

        // rt_fl<4, 4> W1_fl_reg;
        zero(W1_fl_reg);
        mma_AtB(W1_fl_reg, XB_col_reg, Z1_col_reg, W1_fl_reg);

        // rt_bf<4, 4> W1_row_reg;
        copy(W1_row_reg, W1_fl_reg);
        rt_bf<4, 4, kittens::ducks::rt_layout::col> &W1_col_reg = swap_layout_inplace(W1_row_reg);

        sub(W1_reg, W1_reg, W1_col_reg);
    }

    store(_W1, W1_reg, W1_reg.cols);
}


void
prefill_whole_loop
        (
                torch::Tensor W1,
                torch::Tensor XA,
                torch::Tensor XB,
                torch::Tensor XC,
                torch::Tensor Output,
                hipStream_t stream
        ) {
    auto batch = XA.size(0);
    auto head = XA.size(1);
    auto NC = XA.size(2);
    auto CS = XA.size(3);
    auto HF = XA.size(4);

    using H = __hip_bfloat16;
    using T = c10::BFloat16;
    const int workers = 1;

    auto threads = workers * kittens::WARP_THREADS;

    std::cout << "M1 TK whole loop" << std::endl;

    prefill_whole_loop_ker<H, T><<<batch * head, threads, 0, stream>>>(
            NC, CS, HF,
            W1.data_ptr<T>(),
            XA.data_ptr<T>(), XB.data_ptr<T>(), XC.data_ptr<T>(),
            Output.data_ptr<T>()
    );


}

